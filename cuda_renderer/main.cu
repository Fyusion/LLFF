#include <stdio.h>
#include <iostream>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include <helper_functions.h>
// #include <hip/hip_runtime_api.h>

#define GLM_FORCE_CUDA
#define GLM_FORCE_SWIZZLE
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <glm/gtc/matrix_access.hpp> 

#include "mpi_cuda.h"

#include <iostream>
#include <iomanip>

#include <algorithm>
#include <vector>
#include <sstream>
#include <fstream>

#include <chrono> 
using namespace std::chrono; 

using namespace std;
using namespace glm;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



void gpu2ffmpeg(const char* filename, uint8_t* d_out_arr_vid,
                  int imageWidth, int imageHeight, int N_frames, int crf) {
    
    FILE *pPipe;
    long lSize;
    int imgcols = imageWidth, imgrows = imageHeight, elemSize = 3;
    
    stringstream sstm;
    sstm << "/usr/bin/ffmpeg -y -framerate 30 -f rawvideo -vcodec rawvideo -s " << imgcols << "x" << imgrows  
        <<" -pix_fmt rgb24 -i - -pix_fmt yuv420p -r 30 -crf " << crf << " -c:v libx264 -shortest " << filename;

    cout << "ffmpeg, calling:" << endl;
    cout << sstm.str() << endl;
    // open a pipe to FFmpeg
	#ifdef _WIN32
	   if ( !(pPipe = _popen(sstm.str().c_str(), "w")) ) {
	#else
		if ( !(pPipe = popen(sstm.str().c_str(), "w")) ) {
	#endif
        cout << "popen error" << endl;
        exit(1);
    }

    // write to pipe
    lSize = imgrows * imgcols * elemSize;
    uint8_t* out_arr = new uint8_t[lSize];
    
    for (int i = 0; i < N_frames; ++i) {
        hipMemcpy(out_arr, d_out_arr_vid + lSize * i, lSize, hipMemcpyDeviceToHost);
        gpuErrchk(hipDeviceSynchronize());
        fwrite(out_arr, 1, lSize, pPipe);        
    }
    fflush(pPipe);
    fclose(pPipe);
    
    delete [] out_arr;
}


void load_render_poses(const char* posefile, vector<mat4>& poses, int& w, int& h, float& f) {
    float buffer[16];
    
    std::ifstream file(posefile);
    float wf, hf;
    int N_poses=0;
    string line;
    
    int l = 0;
    while (getline(file, line)) {
        // cout << l++ << ": " << line << endl;
        auto sline = istringstream(line);
        if (N_poses==0){
            sline >> N_poses;
            cout << "Loading render poses: " << posefile << ", " << N_poses << endl;
        } else {
            
            for (int j = 0; j < 16; ++j) {
                if (j%4==3) {
                    buffer[j] = j < 15 ? 0.f : 1.f;
                } else {
                    sline >> buffer[j];
                    // cout << buffer[j] << ", ";
                }
            }
            // cout << endl;
            sline >> hf >> wf >> f;
            mat4 p;
            memcpy(value_ptr(p), buffer, 16 * sizeof(float));
            poses.push_back(p);
            // cout << l << ": " << hf << " " << wf << " " << f << endl;
        }
        l++;
    }
    w = (int)wf;
    h = (int)hf;
    cout << w << " x " << h << ", " << f << endl;
    
}

void render_poses(MPIMeta meta, const char* mpidir, const char* posefile, const char* videofile, 
                  int height=-1, float scale=1., int crf=18) {
    cout << "Begin render poses" << endl;
    
    int imageWidth, imageHeight;
    float focal;
    vector<mat4> poses;
    
    load_render_poses(posefile, poses, imageWidth, imageHeight, focal);
    
    if (height <= 0) {
        height = meta.height;
    }
    
    float factor = height / (float)imageHeight;
    imageWidth  = int(imageWidth  * factor * scale * .5) * 2;
    imageHeight = int(imageHeight * factor * scale * .5) * 2;
    focal *= factor;
    
    int N_blend = 5;
    
    int N_frames = poses.size();
    
    // GPU buffer for frames
    uint8_t* d_out_arr_vid;
    unsigned long N_bytes = 3 * imageWidth * imageHeight;
    cout << "Alloc video buffer on GPU " << N_frames * N_bytes / (1<<20) << " MB" << endl;
    gpuErrchk(hipMalloc(&d_out_arr_vid, N_frames * N_bytes));
    
    auto start = high_resolution_clock::now(); 
    
    cout << "Render frames" << endl;
    for (int i = 0; i < N_frames; ++i) {
        
        glm::mat4 pose = poses[i];
        
        meta.render_pose(glm::value_ptr(pose), d_out_arr_vid + N_bytes * i, 
                     imageHeight, imageWidth, focal, N_blend);
        
    }
    gpuErrchk(hipDeviceSynchronize());
    
    auto stop = high_resolution_clock::now(); 
    auto duration = duration_cast<milliseconds>(stop - start); 

    cout << "End render vid " << duration.count() << " ms" << endl;
    
    
    std::cout << "Saving " << videofile << std::endl;
    gpu2ffmpeg(videofile, d_out_arr_vid,
                  imageWidth, imageHeight, N_frames, crf);
    
    cout << "finished" << endl;
    
}



void run(const char* mpidir, const char* posefile, const char* videofile,
                    int height, float scale, int crf) {
    
    MPIMeta meta;
    
    std::cout << "Loading " << mpidir << std::endl;
    meta.load_all_mpis(mpidir);
    
    meta.mpis2gpu();
    
    render_poses(meta, mpidir, posefile, videofile, height, scale, crf);
    
}



int main(int argc, const char* argv[]) {
    
    if (argc < 7) {
        std::cout << "Usage: demo <mpidir> <posefile> <videofile> <height> <scale> <crf>" << std::endl;
        return -1;
    }
    run(argv[1], argv[2], argv[3], stoi(argv[4]), stof(argv[5]), stoi(argv[6]));
    
    cout << "Done!" << endl;
}